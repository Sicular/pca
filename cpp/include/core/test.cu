/*
 * How to compile (assume cuda is installed at /usr/local/cuda/)
 *   nvcc -c -I/usr/local/cuda/include csrms2.cpp
 *   g++ -o csrm2 csrsm2.o -L/usr/local/cuda/lib64 -lcusparse -lcudart
 */
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <stdio.h>
#include <stdlib.h>

#include <iostream>

using namespace std;

__global__ void update(int *d_str) { d_str[0] += 1; }

hipsparseStatus_t create_sparse(hipsparseSpMatDescr_t *spMatDescr,
                               unsigned int *indptr, unsigned int *indices,
                               float *data) {
  int a = 0;
  hipMemcpy(&a, indptr + 2, sizeof(int), hipMemcpyDeviceToHost);
  std::cout << a << std::endl;
  return hipsparseCreateCsr(spMatDescr, 4, 4, 9, indptr, indices, data,
                           HIPSPARSE_INDEX_16U, HIPSPARSE_INDEX_16U,
                           HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F);
}

void print_any_thing(int str) {
  hipsparseSpMatDescr_t descrA;
  unsigned int csrRowPtrA[] = {1, 4, 5, 8, 10};
  unsigned int csrColIndA[] = {1, 3, 4, 2, 1, 3, 4, 2, 4};
  float csrValA[] = {1, 2, -3, 4, 5, 6, 7, 8, 9};
  unsigned int *d_csrRowPtrA = nullptr;
  unsigned int *d_csrColIndA = nullptr;
  float *d_csrValA = nullptr;

  hipMalloc(&d_csrRowPtrA, sizeof(int) * 5000);
  hipMalloc(&d_csrColIndA, sizeof(int) * 9000);
  hipMalloc(&d_csrValA, sizeof(float) * 9000);
  hipMemcpy(d_csrRowPtrA, csrRowPtrA, sizeof(int) * 5, hipMemcpyHostToDevice);
  hipMemcpy(d_csrColIndA, csrColIndA, sizeof(int) * 9, hipMemcpyHostToDevice);
  hipMemcpy(d_csrValA, csrValA, sizeof(float) * 9, hipMemcpyHostToDevice);

  auto stt = create_sparse(&descrA, d_csrRowPtrA, d_csrColIndA, d_csrValA);

  std::cout << (stt == HIPSPARSE_STATUS_SUCCESS) << std::endl;

  int *d_str;
  // int32_t
  // Allocate device memory for a
  // cudaMalloc((void **)&d_str, sizeof(int) * 1);

  // Transfer data from host to device memory
  // cudaMemcpy(d_str, &str, sizeof(int) * 1, cudaMemcpyHostToDevice);
  // update<<<1, 1>>>(d_str);
  // cudaMemcpy(&str, d_str, sizeof(int) * 1, cudaMemcpyDeviceToHost);
  // cudaFree(d_str);
  printf("%d\n", str);
};
